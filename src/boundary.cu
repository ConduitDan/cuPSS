#include "hip/hip_runtime.h"
#include "boundary.h"


BoundaryConditions::BoundaryConditions(BoundaryType type, BoundaryDirection dimension, std::function<float(float,float,float)> value)
    :_type(type),_dimension(dimension),_value_fn(value),_single_value(false){}
     // 3d constructor
BoundaryConditions::BoundaryConditions(BoundaryType type, BoundaryDirection dimension, float value)
    :_type(type),_dimension(dimension),_value(value),_single_value(true){} //2d constructor

void BoundaryConditions::initalize(field * myField){
    // ok first thing we need to do is grab the pointer to the field
    // _field = myEvolver->fieldsMap[_fieldName]; // TO DO Name validation
    _fieldSize = myField->get_size();
    _fieldSpacing = myField->get_spaceing();
    _with_cuda = myField->isCUDA;
    _boundarySize = _fieldSize;
    _boundarySize[_dimension/2] = 1; // we don't need to iterate over the demension we're setting the boundary on


    // now that we have that we have dimensional information, we need to check to see if we need to calculate values for the boundary (if its not a single value)
    if (!_single_value) {
        // if we have to do that allocate the space here, figure out the values and then transfer them to the GPU if we're using it
        // the numebr of values we needs is the product of the dimensions that aren't this one
        
        long boundarySize = 1;
        for (long i = 0; i<3; i++){
           boundarySize*=_boundarySize[i];
        }
        _values = std::unique_ptr<float[]>(new float[_fieldSize[boundarySize]]);
        
        long index = 0;
        std::array<float,3> position;
        for (iz = 0; iz<_boundarySize[2]; iz++){
            for (iy = 0; iy<_boundarySize[1]; iy++){
                for (ix = 0; i<_boundarySize[0]; ix++){
                    position = {ix*_fieldSpacing[0],iy*_fieldSpacing[1],iz*_fieldSpacing[2]};
                    // if we're on the right boundary we need to correct the value of that position
                    if (dimension%2 == 1) {
                        // right boundary 
                        position[dimension/2]=(_fieldSize[dimension/2]-1)*_fieldSpacing[dimension/2];// hmmm might need to think carefully about how to encorpurate boundary layers here. for now ignore
                    }
                    _values[index] = _value_fn(position[0],position[1],position[2]);
                    index++;
                }
            }
        }
        
        if (_with_cuda){
            _threadDim=(32,32,32);
            switch (_dimension/2){
                case 0:
                    threadDim.x = 1;
                    break;
                case 1:
                    threadDim.y = 1;
                    break;
                case 2:
                    threadDim.z = 1;
                    break;
            }
            bx = (_boundarySize[0]+_threadDim.x-1)/_threadDim.x;
            by = (_boundarySize[1]+_threadDim.y-1)/_threadDim.y;
            bz = (_boundarySize[2]+_threadDim.z-1)/_threadDim.z;
            _blockDim = dim3(bx,by,bz);

            hipMalloc(reinterpret_cast<void **>(&d_values), boundarySize * sizeof(float));
            hipMemcpy(d_values, _values.get(), boundarySize * sizeof(float), hipMemcpyHostToDevice);
        }
    }
}
long BoundaryConditions::flatten_index(std::array<int,3> dimension_index)
{
    // index = xi + nx *yi +nx *ny *zi 
    return dimension_index[0]+dimension_index[1]*_fieldSize[0]+dimension_index[2]*_fieldSize[0]*_fieldSize[1];
}
void BoundaryConditions::operator(float2* fieldValues)
{
    switch (_type){
        case BoundaryType::Dirichlet:
            applyDirichelt(fieldValues);
            break;
        case BoundaryType::VonNeumann:
            applyVonNeumann(fieldValues);
            break;

    }
} 
void BoundaryConditions::applyDirichelt(float2* fieldValues) 
{
    if (_with_cuda)
    {
        bool leftwall = !(_dimension%2)
        dim3 field_size = dim3(_fieldSize[0], _fieldSize[1], _fieldSize[2])
        dim3 boundary_size = dim3(_boundarySize[0], _boundarySize[1], _boundarySize[2])

        if (_single_value) 
        {
            applyDiricheltSingleValue_gpu(fieldValues,_value,_depth,_dimension/2, leftwall, field_size,  boundary_size,  _blockDim,  _threadDim);
        }
        else {
            applyDiricheltMultipleValue_gpu(fieldValues,_values,_depth,_dimension/2, leftwall, field_size,  boundary_size,  _blockDim,  _threadDim);
        }

    } 
    
    else 
    {
        long valueIndex = 0;
        long fieldIndex = 0;
        std::array<int,3> dimension_index;
        for (iz = 0; iz<_boundarySize[2]; iz++) {
            for (iy = 0; iy<_boundarySize[1]; iy++) {
                for (ix = 0; i<_boundarySize[0]; ix++) {
                    for (ib = 0; ib < _depth; ib ++) {
                        dimension_index = {ix,iy,iz};
                        if (dimension%2 == 0) {
                            // left wall
                            dimension_index[dimension/2]=ib;
                        }
                        if (dimension%1 == 1){
                            // right wall
                            dimension_index[dimension/2]=_fieldSize[dimension/2]-ib-1;
                        }

                        fieldIndex = flatten_index(dimension_index);
                        if (_single_value){// add aliased array
                            fieldValues[fieldIndex].x=_value;
                        } else {
                            fieldValues[fieldIndex].x=_values[valueIndex];
                        }

                    }
                    valueIndex++;
                }
            }
        }
    }
}
void BoundaryConditions::applyVonNeumann(float2* fieldValues){
    if (_with_cuda)
    {
        bool leftwall = !(_dimension%2)
        dim3 field_size = dim3(_fieldSize[0], _fieldSize[1], _fieldSize[2])
        dim3 boundary_size = dim3(_boundarySize[0], _boundarySize[1], _boundarySize[2])
        float h = _fieldSpacing[_dimension/2];
        if (_single_value) 
        {
            applyVonNuemannSingleValue_gpu(fieldValues,_value,_depth,_dimension/2, leftwall, field_size,  boundary_size,h,  _blockDim,  _threadDim);
        }
        else {
            applyVonNuemannMultipleValue_gpu(fieldValues,_values,_depth,_dimension/2, leftwall, field_size,  boundary_size, h ,  _blockDim,  _threadDim);
        }

    } else {
        long valueIndex = 0;
        long fieldIndex = 0;
        long fieldIndexOneIn = 0;
        std::array<int,3> dimension_index;
        std::array<int,3> dimension_index_one_in;
        for (iz = 0; iz<_boundarySize[2]; iz++) {
            for (iy = 0; iy<_boundarySize[1]; iy++) {
                for (ix = 0; i<_boundarySize[0]; ix++) {
                    for (ib = 0; ib < _depth; ib ++) {
                        dimension_index = {ix,iy,iz};
                        dimension_index_one_in=dimension_index;
                        // for von nueman we have to start from the inside and go out to keep the deriviate correct
                        if (dimension%2 == 0) {
                            // left wall
                            dimension_index[dimension/2]=(_depth-ib-1);
                            dimension_index_one_in[dimension/2]=(_depth-ib);

                        }
                        if (dimension%1 == 1){
                            // right wall
                            dimension_index[dimension/2]=_fieldSize[dimension/2]-1-(ib-_depth-1);
                            dimension_index_one_in[dimension/2]=fieldSize[dimension/2]-1-(ib-_depth);

                        }

                        fieldIndex = flatten_index(dimension_index);
                        fieldIndexOneIn = flatten_index(dimension_index_one_in);
                        // (x[one in]-x)/dx = value
                        // x = x[one in] - dx *value;
                        if (_single_value){
                            fieldValues[fieldIndex]=fieldValues[fieldIndexOneIn]-_fieldSpacing[dimension/2]*_value;
                        } else {
                            fieldValues[fieldIndex]=fieldValues[fieldIndexOneIn]-_fieldSpacing[dimension/2]*_values[valueIndex];
                        }

                    }
                    valueIndex++;
                }
            }
        }
    }
}