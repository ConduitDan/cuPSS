#include "hip/hip_runtime.h"
#include <cmath>
#include <hip/driver_types.h>
#include <iostream>
#include <cstdlib>
#include <iomanip>
#include <fstream>
#include <ostream>
#include <string>
#include <hip/hip_runtime.h>
#include <sys/stat.h>
#include "../inc/cupss.h"
#include "evolver.h"

void evolver::common_constructor()
{
    currentTime = 0.0f;
    currentTimeStep = 0;
    dtsqrt = std::sqrt(dt);

    if (sz == 1)
    {
        if (sy == 1)
        {
            dimension = 1;
            blocks = 1;
            threads_per_block = sx;
        }
        else 
        {
            dimension = 2;
            threads_per_block = dim3(32,32);
            int bx = (sx+31)/32;
            int by = (sy+31)/32;
            // blocks = dim3(sx/32,sy/32);
            blocks = dim3(bx,by);
        }
    }
    else 
    {
        dimension = 3;
        threads_per_block = dim3(16, 8, 8);
        int bx = (sx+15)/16;
        int by = (sy+7)/8;
        int bz = (sz+7)/8;

        blocks = dim3(bx,by,bz);
    }

    writePrecision = 6;
    _parser = new parser(this);
}

evolver::evolver(bool _with_cuda, int _sx, float _dx, float _dt, int _ses) : sx(_sx), sy(1), sz(1), dx(_dx), dy(1.0f), dz(1.0f), dt(_dt), writeEveryNSteps(_ses)
{
    std::srand(time(0));
    with_cuda = _with_cuda;
    
    common_constructor();
}

evolver::evolver(bool _with_cuda, int _sx, int _sy, float _dx, float _dy, float _dt, int _ses) : sx(_sx), sy(_sy), sz(1), dx(_dx), dy(_dy), dz(1.0f), dt(_dt), writeEveryNSteps(_ses)
{
    std::srand(time(NULL));
    with_cuda = _with_cuda;

    common_constructor();
}

evolver::evolver(bool _with_cuda, int _sx, int _sy, int _sz, float _dx, float _dy, float _dz, float _dt, int _ses) : sx(_sx), sy(_sy), sz(_sz), dx(_dx), dy(_dy), dz(_dz), dt(_dt), writeEveryNSteps(_ses)
{
    std::srand(time(NULL));
    with_cuda = _with_cuda;

    common_constructor();
}

int evolver::createFromFile(const std::string &file)
{
    _parser->createFromFile(file);
    return 0;
}

void evolver::prepareProblem()
{
    struct stat info;
    std::string pathname = "data";

    if ( stat( pathname.c_str(), &info ) != 0 )
    {
        std::cout << "data directory not found, creating it.\n";
        int dir_err = mkdir(pathname.c_str(), S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);
        if (dir_err == -1) {
            std::cout << "Error creating data directory\n";
            std::exit(1);
        }
    }
    else if ( info.st_mode & S_IFDIR )
    {
        // data directory already exists, do nothing
    }
    else 
    {
        std::cout << "Can't create data directory, is there a file called data?\n";
        std::exit(1);
    }
    // copy host to device to account for initial conditions
    _parser->writeParamsToFile("data/parameter_list.txt");

    for (int i = 0; i < fields.size(); i++)
    {
        fields[i]->copyHostToDevice();
        fields[i]->toComp();
    }
    // for each field prepare device and precalculate implicits
    for (int i = 0; i < fields.size(); i++)
    {
        fields[i]->prepareDevice();
        fields[i]->precalculateImplicit(dt);
        fields[i]->system_p = this;
    }
}

void evolver::setOutputField(std::string _name, int _output)
{
    for (int i = 0; i < fields.size(); i++)
    {
        if (fields[i]->name == _name)
        {
            if (_output)
                fields[i]->outputToFile = true;
            else
                fields[i]->outputToFile = false;
            return;
        }
    }
    std::cout << "setOutputField EROR: " << _name << " not found." << std::endl;
}

int evolver::addParameter(std::string _name, float value)
{
    _parser->insert_parameter(_name, value);
    return 0;
}

int evolver::addEquation(std::string equation)
{
    _parser->add_equation(equation);
    return 0;
}

int evolver::addBoundaryCondition(std::string _name,BoundaryConditions BC)
{
    int fieldIndex = existsField(_name);
    if (fieldIndex != -1)
        fields[fieldIndex]->addBoundaryCondition(BC);
    
}
int evolver::existsField(std::string _name)
{
    int foundIndex = -1;
    for (int i = 0; i < fields.size(); i++)
    {
        if (fields[i]->name == _name)
        {
            foundIndex = i;
            return foundIndex;
        }
    }
    return foundIndex;
}

int evolver::addNoise(std::string _name, std::string equation)
{
    if (existsField(_name) == -1)
    {
        std::cout << "Adding noise to non existing field! (" << _name << ")" << std::endl;
        return -1;
    }
    pres prefactor = _parser->add_noise(equation);
    fieldsMap[_name]->isNoisy = true;
    fieldsMap[_name]->noise_amplitude = prefactor;
    return 0;
}

void evolver::addField(field *newField)
{
    fields.push_back(newField);
}

int evolver::createField(std::string name, bool dynamic)
{
    for (int i = 0; i < fields.size(); i++)
    {
        if (fields[i]->name == name)
        {
            std::cout << "Trying to create field with name that already exists" << std::endl;
            return 1;
        }
    }
    field *newField = new field(sx, sy, sz, dx, dy, dz);
    newField->name = name;
    newField->isCUDA = with_cuda;
    newField->dynamic = dynamic;
    newField->blocks = blocks;
    newField->threads_per_block = threads_per_block;
    fields.push_back(newField);

    fieldsMap[name] = fields[fields.size()-1];
    fieldsReal[name] = fields[fields.size()-1]->real_array;
    fieldsFourier[name] = fields[fields.size()-1]->comp_array;
    return 0;
}


int evolver::advanceTime()
{
    if (currentTimeStep % writeEveryNSteps == 0)
    {
        // Maybe calculate observables and write them out
        writeOut();
    }
    // Loop over each field 
        // Calculate RHSs
    for (int i = 0; i < fields.size(); i++)
    {
        if (!fields[i]->dynamic)
            fields[i]->updateTerms();
    }
    for (int i = 0; i < fields.size(); i++)
    {
        if (!fields[i]->dynamic)
            fields[i]->setRHS(dt); 
    }
    for (int i = 0; i < fields.size(); i++)
    {
        if (fields[i]->dynamic)
            fields[i]->updateTerms();
    }
    for (int i = 0; i < fields.size(); i++)
    {
        if (fields[i]->dynamic)
            fields[i]->setRHS(dt); 
    }

    currentTime += dt;
    currentTimeStep += 1;
    return 0;
}

void evolver::writeOut()
{
    for (int f = 0; f < fields.size(); f++)
    {
        fields[f]->writeToFile(currentTimeStep, dimension, writePrecision);
    }
}

void evolver::printInformation()
{
    std::cout << std::fixed;
    std::cout << std::setprecision(3);
    std::cout << "Information on this evolver:" << std::endl;
    std::cout << dimension << "-dimensional system of size " << sx << "x" << sy << "x" << sz << std::endl;
    std::cout << "Physical size " << (float)sx*dx << "x"
        << (float)sy*dy << "x" << (float)sz*dz << " with cells of size " 
        << dx << "x" << dy << "x" << dz << std::endl;
    std::cout << "There are " << fields.size() << " fields." << std::endl;
    for (int i = 0; i < fields.size(); i++)
    {
        std::cout << "Field " << i << ": " << fields[i]->name;
        if (fields[i]->dynamic) std::cout << " is dynamic.";
        else std::cout << " is not dynamic";
        std::cout << " and has " << fields[i]->terms.size() << " explicit terms";
        std::cout << " and " << fields[i]->implicit.size() << " implicit terms.";
        std::cout << " Runs on GPU: " << fields[i]->isCUDA;
        if (fields[i]->needsaliasing)
            std::cout << ". Will be dealiased for a nonlinearity of order " << fields[i]->aliasing_order;
        else
            std::cout << ". Will not be dealised.";
        std::cout << std::endl << "\t";
        if (fields[i]->dynamic) std::cout << "(d/dt)";
        std::cout << fields[i]->name;
        if (fields[i]->dynamic) std::cout << " = ";

        if (fields[i]->implicit.size() > 0)
        {
            std::string implicitLine = "[";
            for (int j = 0; j < fields[i]->implicit.size(); j++)
            {
                float pre = fields[i]->implicit[j].preFactor;
                if (pre > 0.0f) implicitLine += "+";
                implicitLine += std::to_string(pre);
                if (fields[i]->implicit[j].iqx != 0)
                    implicitLine += "(iqx)^(" + std::to_string(fields[i]->implicit[j].iqx) + ")";
                if (fields[i]->implicit[j].iqy != 0)
                    implicitLine += "(iqy)^(" + std::to_string(fields[i]->implicit[j].iqy) + ")";
                if (fields[i]->implicit[j].iqz != 0)
                    implicitLine += "(iqz)^(" + std::to_string(fields[i]->implicit[j].iqz) + ")";
                if (fields[i]->implicit[j].q2n != 0)
                    implicitLine += "(q)^(" + std::to_string(2*fields[i]->implicit[j].q2n) + ")";
                if (fields[i]->implicit[j].invq != 0)
                    implicitLine += "(1/|q|)^(" + std::to_string(fields[i]->implicit[j].invq) + ")";
            }
            implicitLine += "]";
            if (fields[i]->dynamic) implicitLine += fields[i]->name;
            std::cout << implicitLine;
        }
        if (!fields[i]->dynamic)
            std::cout << " = ";
        for (int j = 0; j < fields[i]->terms.size(); j++)
        {
            std::string line = "";
            if(j != 0) line = " + [";
            else line = " [";
            for (int p = 0; p < fields[i]->terms[j]->prefactors_h.size(); p++)
            {
                float pre = fields[i]->terms[j]->prefactors_h[p].preFactor;
                line += " + (" + std::to_string(pre) + ")";
                if (fields[i]->terms[j]->prefactors_h[p].iqx != 0)
                    line += "(iqx)^(" + std::to_string(fields[i]->terms[j]->prefactors_h[p].iqx) + ")";
                if (fields[i]->terms[j]->prefactors_h[p].iqy != 0)
                    line += "(iqy)^(" + std::to_string(fields[i]->terms[j]->prefactors_h[p].iqy) + ")";
                if (fields[i]->terms[j]->prefactors_h[p].iqz != 0)
                    line += "(iqz)^(" + std::to_string(fields[i]->terms[j]->prefactors_h[p].iqz) + ")";
                if (fields[i]->terms[j]->prefactors_h[p].q2n != 0)
                    line += "(q)^(" + std::to_string(2*fields[i]->terms[j]->prefactors_h[p].q2n) + ")";
                if (fields[i]->terms[j]->prefactors_h[p].invq != 0)
                    line += "(1/|q|)^(" + std::to_string(fields[i]->terms[j]->prefactors_h[p].invq) + ")";
                if (p != fields[i]->terms[j]->prefactors_h.size()-1)
                    line += " + ";
            }
            line += "] ";
            line += "(";
            for (int k = 0; k < fields[i]->terms[j]->product.size(); k++)
                line += " " + fields[i]->terms[j]->product[k]->name;
            line += " )";
            std::cout << line;
        }
        if (fields[i]->isNoisy)
        {
            std::cout << "+ sqrt[2*" << fields[i]->noise_amplitude.preFactor;
            if (fields[i]->noise_amplitude.q2n != 0)
                std::cout << "*q^" << fields[i]->noise_amplitude.q2n * 2;
            if (fields[i]->noise_amplitude.invq != 0)
                std::cout << "*1/|q|^" << fields[i]->noise_amplitude.q2n;
            // print amplitude function
            std::cout << "] x noise";
        }
        std::cout << std::endl << std::endl;
    }
}

int evolver::createTerm(std::string _field, const std::vector<pres> &_prefactors, const std::vector<std::string> &_product)
{
    int field_index = -1;

    for (int i = 0; i < fields.size(); i++)
    {
        if (fields[i]->name == _field)
        {
            field_index = i;
            break;
        }
    }

    if (field_index == -1)
    {
        std::cout << "Field " << _field << " not found trying to create term" << std::endl;
        return 1;
    }

    term *newTerm = new term(sx, sy, sz, dx, dy, dz);
    newTerm->isCUDA = with_cuda;

    for (int i = 0; i < _product.size(); i++)
    {
        std::string fieldForProduct = _product[i];
        for (int j = 0; j < fields.size(); j++)
        {
            if (fieldForProduct == fields[j]->name)
            {
                newTerm->product.push_back(fields[j]);
            }
        }
    }

    for (int i = 0; i < _prefactors.size(); i++)
    {
        newTerm->prefactors_h.push_back(_prefactors[i]);
    }

    newTerm->blocks = blocks;
    newTerm->threads_per_block = threads_per_block;

    fields[field_index]->terms.push_back(newTerm);
    return 0;
}

void evolver::copyAllDataToHost()
{
    for (int i = 0; i < fields.size(); i++)
    {
        fields[i]->copyDeviceToHost();
    }
}

void evolver::initializeUniform(std::string field, float value)
{
    bool found = false;
    for (int i = 0; i < fields.size(); i++)
        if (field == fields[i]->name)
            found = true;
    if (!found)
    {
       std::cout << "ERROR in initialize uniform, " << field << " not found" << std::endl;
       std::exit(1);
    }
    for (int k = 0; k < sz; k++)
    {
        for (int j = 0; j < sy; j++)
        {
            for (int i = 0; i < sx; i++)
            {
                int index = k * sx * sy + j * sx + i;
                fieldsMap[field]->real_array[index].x = value;
            }
        }
    }
}

void evolver::initializeUniformNoise(std::string field, float value)
{
    bool found = false;
    for (int i = 0; i < fields.size(); i++)
        if (field == fields[i]->name)
            found = true;
    if (!found)
    {
        std::cout << "ERROR in initialize uniform, " << field << " not found" << std::endl;
        std::exit(1);
    }
    srand(time(0));
    for (int k = 0; k < sz; k++)
    {
        for (int j = 0; j < sy; j++)
        {
            for (int i = 0; i < sx; i++)
            {
                int index = k * sx * sy + j * sx + i;
                fieldsMap[field]->real_array[index].x = value * 0.01f * (float)(rand()%200-100);
            }
        }
    }
}

void evolver::initializeNormalNoise(std::string field, float mean, float sigma)
{
    bool found = false;
    for (int i = 0; i < fields.size(); i++)
        if (field == fields[i]->name)
            found = true;
    if (!found)
    {
        std::cout << "ERROR in initialize uniform, " << field << " not found" << std::endl;
        std::exit(1);
    }
    srand(time(0));
    float v1 = 0.0;
    float v2 = 0.0;
    for (int k = 0; k < sz; k++)
    {
        for (int j = 0; j < sy; j++)
        {
            for (int i = 0; i < sx; i++)
            {
                int index = k * sx * sy + j * sx + i;
                // Use Box-Muller algorithm
                if (index%2 == 0)
                {
                    v1 = 0.01*(float)(rand()%100+1);
                    v2 = 0.01*(float)(rand()%100+1);
                    fieldsMap[field]->real_array[index].x = sigma*sigma*std::sqrt(-2.0*std::log(v1))*std::cos(2.0*PI*v2) + mean;
                    fieldsMap[field]->real_array[index+1].x = sigma*sigma*std::sqrt(-2.0*std::log(v1))*std::sin(2.0*PI*v2) + mean;
                }
            }
        }
    }
}

void evolver::initializeHalfSystem(std::string field, float val1, float val2, float xi, int direction)
{
    bool found = false;
    for (int i = 0; i < fields.size(); i++)
        if (field == fields[i]->name)
            found = true;
    if (!found)
    {
        std::cout << "ERROR in initialize half system, " << field << " not found" << std::endl;
        std::exit(1);
    }
    if (xi <= 0.0)
    {
        std::cout << "ERROR in initialize, interface width cannot be 0 or negative" << std::endl;
        std::exit(1);
    }
    if (direction < 1 || direction > 3)
    {
        std::cout << "ERROR in initialize, direction can be 1, 2 or 3 for x, y, z, respectively" << std::endl;
        std::exit(1);
    }
    srand(time(0));
    int rev_size = sx;
    if (direction == 2)
        rev_size = sy;
    if (direction == 3)
        rev_size = sz;
    for (int k = 0; k < sz; k++)
    {
        for (int j = 0; j < sy; j++)
        {
            for (int i = 0; i < sx; i++)
            {
                int index = k * sx * sy + j * sx + i;
                int ref = i;
                if (direction == 2)
                    ref = j;
                if (direction == 3)
                    ref = k;
                fieldsMap[field]->real_array[index].x = val1 + (val2-val1)*0.5*(1.0 + std::tanh((ref - rev_size/2)/(std::sqrt(2)*xi)));
            }
        }
    }
}

void evolver::initializeDroplet(std::string field, float val1, float val2, float radius, float xi, int p_x, int p_y, int p_z)
{
    bool found = false;
    for (int i = 0; i < fields.size(); i++)
        if (field == fields[i]->name)
            found = true;
    if (!found)
    {
        std::cout << "ERROR in initialize droplet, " << field << " not found" << std::endl;
        std::exit(1);
    }
    if (xi <= 0.0 || radius <= 0.0)
    {
        std::cout << "ERROR in initialize, droplet radius and interface width cannot be 0 or negative" << std::endl;
        std::exit(1);
    }
    int x_ = p_x % sx;
    int y_ = p_y % sy;
    int z_ = p_z % sz;
    srand(time(0));
    for (int k = 0; k < sz; k++)
    {
        for (int j = 0; j < sy; j++)
        {
            for (int i = 0; i < sx; i++)
            {
                int index = k * sx * sy + j * sx + i;
                float x_d = i - x_;
                float y_d = j - y_;
                float z_d = k - z_;
                float r_c = std::sqrt(x_d*x_d + y_d*y_d + z_d*z_d);
                fieldsMap[field]->real_array[index].x = val1 + (val2-val1)*0.5*(1.0 + std::tanh((r_c - radius)/(std::sqrt(2)*xi)));
            }
        }
    }
}

int evolver::getSystemSizeX()
{
    return sx;
}
int evolver::getSystemSizeY()
{
    return sy;
}
int evolver::getSystemSizeZ()
{
    return sz;
}
float evolver::getSystemPhysicalSizeX()
{
    return ((float)sx)*dx;
}
float evolver::getSystemPhysicalSizeY()
{
    return ((float)sy)*dy;
}
float evolver::getSystemPhysicalSizeZ()
{
    return ((float)sz)*dz;
}
